
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void SomaVetores(int* vetorA, int* vetorB, int* vetorC, int tamanho)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < tamanho)
        vetorC[i] = vetorA[i] + vetorB[i];
}

int main()
{
    int tamanho = 100000000;
    size_t totalBytes = tamanho * sizeof(int);
    
    int* vetorA = (int*) malloc(totalBytes);
    int* vetorB = (int*) malloc(totalBytes);
    int* vetorC = (int*) malloc(totalBytes);

    
    if(vetorA == NULL || vetorB == NULL || vetorC == NULL)
    {
        cout << "Memoria insuficiente!" << endl;
        return 0;
    }

    for(int index = 0; index < tamanho; index++)
    {
        vetorA[index] = vetorB[index] = index; 
        vetorC[index] = 0; 
    }

    int* cudaVetorA;
    int* cudaVetorB;
    int* cudaVetorC;
    
    hipMalloc(&cudaVetorA, totalBytes);
    hipMalloc(&cudaVetorB, totalBytes);
    hipMalloc(&cudaVetorC, totalBytes);

    hipMemcpy(cudaVetorA, vetorA, totalBytes, hipMemcpyHostToDevice);
    hipMemcpy(cudaVetorB, vetorB, totalBytes, hipMemcpyHostToDevice);

    SomaVetores<<<1, tamanho>>>(cudaVetorA, cudaVetorB, cudaVetorC, tamanho);

    hipMemcpy(vetorC, cudaVetorC, totalBytes, hipMemcpyDeviceToHost);
   
    hipFree(cudaVetorA);
    hipFree(cudaVetorB);
    hipFree(cudaVetorC);

    /*
    for(int index = 0; index < tamanho; index++)
    {
        cout << "C = " << vetorC[index] << endl;
    }
    */

    free(vetorA);
    free(vetorB);
    free(vetorC);

    cout << "200 OK" << endl;

    return 0;
}
